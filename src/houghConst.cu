#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

// Memoria constante para senos y cosenos
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

// GPU kernel
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale) {
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h) return;

    int xCent = w / 2;
    int yCent = h / 2;

    int xCoord = gloID % w - xCent;
    int yCoord = yCent - gloID / w;

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            // Se utiliza memoria constante para calculos trigoonometricos
            float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            atomicAdd(&acc[rIdx * degreeBins + tIdx], 1);
        }
    }
}

int main(int argc, char **argv) {
    if (argc < 2) {
        printf("Usage: %s <input_image.pgm>\n", argv[0]);
        return -1;
    }

    PGMImage inImg(argv[1]);
    int w = inImg.x_dim;
    int h = inImg.y_dim;

    float *pcCos = (float *) malloc(sizeof(float) * degreeBins);
    float *pcSin = (float *) malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (int i = 0; i < degreeBins; i++) {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    // valores trigonométricos a memoria constante
    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

    unsigned char *d_in;
    int *d_hough;
    int *h_hough = (int *) malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void **) &d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void **) &d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, inImg.pixels, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    // Medición de tiempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int blockNum = ceil((float)(w * h) / 256);
    GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel Execution Time: %f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    // Validación simple con la CPU
    int *cpuht;
    CPU_HoughTran(inImg.pixels, w, h, &cpuht);
    for (int i = 0; i < degreeBins * rBins; i++) {
        if (cpuht[i] != h_hough[i]) {
            printf("Calculation mismatch at: %i %i %i\n", i, cpuht[i], h_hough[i]);
        }
    }
    printf("Done!\n");

    // Liberar memoria
    hipFree(d_in);
    hipFree(d_hough);
    free(h_hough);
    free(pcCos);
    free(pcSin);

    return 0;
}
